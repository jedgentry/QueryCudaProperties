
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

void error() {
	printf("Encountered an error...");
	exit(1);
}

int main()
{
	hipDeviceProp_t prop;
	int count = 0;
	if (hipGetDeviceCount(&count))
		error();
	for (int i = 0; i < count; ++i) {
		if (hipGetDeviceProperties(&prop, i))
			error();
		std::cout << "\tProperties of device #" << i << std::endl;
		std::cout << "Name: " << prop.name << std::endl;
		std::cout << "Compute: " << prop.major << ", " << prop.minor << std::endl;   
		std::cout << "Clock rate: " << prop.clockRate << std::endl;
		std::cout << "Device Overlap: " << prop.deviceOverlap << std::endl;
		std::cout << "Kernel Execution Timeout: " << prop.kernelExecTimeoutEnabled << std::endl;
		std::cout << std::endl << "\tMemory Information" << std::endl;
		std::cout << "Total global memory: " << prop.totalGlobalMem << std::endl;
		std::cout << "Total constant memory: " << prop.totalConstMem << std::endl;
		std::cout << "Max memory pitch: " << prop.memPitch << std::endl;
		std::cout << "Texture Alignment: " << prop.textureAlignment << std::endl;
		std::cout << std::endl << "\tMulti-Processor Information" << std::endl;
		std::cout << "Multiprocessor count: " << prop.multiProcessorCount << std::endl;
		std::cout << "Shared memory per block: " << prop.sharedMemPerBlock << std::endl;
		std::cout << "Registers per multiprocessor: " << prop.regsPerBlock << std::endl;
		std::cout << "Threads of warp: " << prop.warpSize << std::endl;
		std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
		std::cout << "Max thread dimensions: " << prop.maxThreadsDim[0] << ", " <<
			prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << std::endl;
		std::cout << "Max grid dimensions: " << prop.maxGridSize[0] << ", " <<
			prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << std::endl;
		std::cout << std::endl << std::endl;
	}
	return 0;
}